#include "hip/hip_runtime.h"
#include <math.h>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <npp.h>
//#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define DESCALE(x, n)    (((x) + (1 << ((n)-1)))>>(n))
#define COEFFS_0 		(22987)
#define COEFFS_1 		(-11698)
#define COEFFS_2 		(-5636)
#define COEFFS_3 		(29049)
#define clip(minv, maxv, value)  ((value)<minv) ? minv : (((value)>maxv) ? maxv : (value))

__global__ void kernel_yuyv2bgr_(
	unsigned char *dst, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= (width>>1) || y >= height)
		return;

	{
		int Y1, Y2, U, V;
		int r, g, b;

		int si = y*width*2 + x*4;
		int di = y*width*3 + x*6;

		Y1 =  src[si+0];
		U = src[si+1];
		Y2 =  src[si+2];
		V = src[si+3];


		b = DESCALE((U - 128)*COEFFS_3, 14);
		g = DESCALE((U - 128)*COEFFS_2 + (V - 128)*COEFFS_1, 14);
		r = DESCALE((V - 128)*COEFFS_0, 14);

		dst[di+0] = clip(0, 255, Y1 + b);//B
		dst[di+1] = clip(0, 255, Y1 + g);//G
		dst[di+2] = clip(0, 255, Y1 + r);//R
		dst[di+3] = clip(0, 255, Y2 + b);//B
		dst[di+4] = clip(0, 255, Y2 + g);//G
		dst[di+5] = clip(0, 255, Y2 + r);//R
	}
}

extern "C" int yuyv2bgr(
	unsigned char *dst, const unsigned char *src,
	int width, int height,hipStream_t stream)
{
	//dim3 block((width/2+15)/16, (height+63)/64);
	//dim3 thread(16, 64);
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);
	//dim3 block((width/2+127)/128,(height+127)/128);
	//dim3 thread(128, 128);

	//kernel_uyvy2bgr_and_sphere_tp_erect<<<block, thread>>>(dst, src, width, height);
	kernel_yuyv2bgr_<<<block, thread, 0,stream>>>(dst, src, width, height);

	return 0;
}

__global__ void kernel_yuyv2I420(
	unsigned char *dsty, unsigned char *dstu, unsigned char *dstv, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int uvwidth = (width>>1);

	if(x >= (width>>1) || y >= height)
		return;

	dsty[y*width + 2*x] 		= 		src[y*width*2 + x*4];
	dstu[y*uvwidth + x] 	= 		src[y*width*2 + x*4+1];
	dsty[y*width + 2*x+1] = 		src[y*width*2 + x*4+2];
	dstv[y*uvwidth + x] 		= 		src[y*width*2 + x*4+3];
}

extern "C" int yuyv2yuvI420_(
	unsigned char *dsty, unsigned char *dstu,unsigned char *dstv,const unsigned char *src,
	int width, int height,hipStream_t stream)
{
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);

	kernel_yuyv2I420<<<block, thread, 0,stream>>>(dsty,dstu,dstv, src, width, height);

	return 0;
}
__global__ void kernel_yuyv2bgr_I420(
	unsigned char *dstbgr, unsigned char *dsty, unsigned char *dstu,unsigned char *dstv,const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= (width>>2) || y >= height)
		return;

	{
		int Y1, Y2, Y3, Y4, U1, U2, V1, V2 ;
		int r1, g1, b1, r2, g2, b2;

		int si = y*width*2 + x*8;
		int di = y*width*3 + x*12;

		Y1 =  src[si+0];
		U1 = src[si+1];
		Y2 =  src[si+2];
		V1= src[si+3];
		Y3 =  src[si+4];
		U2 = src[si+5];
		Y4 =  src[si+6];
		V2 = src[si+7];

		b1 = DESCALE((U1 - 128)*COEFFS_3, 14);
		g1 = DESCALE((U1 - 128)*COEFFS_2 + (V1 - 128)*COEFFS_1, 14);
		r1 = DESCALE((V1 - 128)*COEFFS_0, 14);

		b2 = DESCALE((U2 - 128)*COEFFS_3, 14);
		g2 = DESCALE((U2 - 128)*COEFFS_2 + (V2 - 128)*COEFFS_1, 14);
		r2 = DESCALE((V2 - 128)*COEFFS_0, 14);

		dstbgr[di+0] = clip(0, 255, Y1 + b1);//B
		dstbgr[di+1] = clip(0, 255, Y1 + g1);//G
		dstbgr[di+2] = clip(0, 255, Y1 + r1);//R
		dstbgr[di+3] = clip(0, 255, Y2 + b1);//B
		dstbgr[di+4] = clip(0, 255, Y2 + g1);//G
		dstbgr[di+5] = clip(0, 255, Y2 + r1);//R

		dstbgr[di+6] = clip(0, 255, Y3 + b2);//B
		dstbgr[di+7] = clip(0, 255, Y3 + g2);//G
		dstbgr[di+8] = clip(0, 255, Y3 + r2);//R
		dstbgr[di+9] = clip(0, 255, Y4 + b2);//B
		dstbgr[di+10] = clip(0, 255, Y4 + g2);//G
		dstbgr[di+11] = clip(0, 255, Y4 + r2);//R

		int dstystride = y*width;
		int dstuvstride = dstystride>>1;

		dsty[dstystride + 4*x] 		= 		Y1;
		dsty[dstystride + 4*x+1] 	= 		Y2;
		dsty[dstystride + 4*x+2] 	= 		Y3;
		dsty[dstystride + 4*x+3] 	= 		Y4;
		dstu[dstuvstride + x] 			= 		(U1+U2)*0.5;
		dstv[dstuvstride + x] 			= 		(V1+V2)*0.5;
	}
}

extern "C" int yuyv2yuvBGRI420_(
		unsigned char *dstrgb, unsigned char *dsty, unsigned char *dstu,unsigned char *dstv,const unsigned char *src,
	int width, int height,hipStream_t stream)
{
	dim3 block((width/4+31)/32,(height+31)/32);
	dim3 thread(32, 32);

	kernel_yuyv2bgr_I420<<<block, thread, 0,stream>>>(dstrgb, dsty,dstu,dstv, src, width, height);

	return 0;
}
hipStream_t streamBGR[4];
unsigned char *tempRGB;
unsigned char *tempYUYV;
unsigned char *tempY;
unsigned char *tempU;
unsigned char *tempV;
unsigned char *pRGALockMem = NULL;
void cuinit()
{
	int w=1920;
	int h=1080;
	for(int i=0; i<4; i++)
		hipStreamCreate(&streamBGR[i]);
	hipMalloc(&tempRGB,w*h*3);
	hipMalloc(&tempYUYV,w*h*2);
	hipMalloc(&tempY,w*h);
	hipMalloc(&tempU,w*h/4);
	hipMalloc(&tempV,w*h/4);

	if(pRGALockMem == NULL){
		int ret = hipHostAlloc((void**)&pRGALockMem, w*h*3, hipHostMallocDefault);
		assert(ret == hipSuccess);
	}
}

void yuyv2BGRI420(cv::Mat& yuyv,cv::Mat& BGR,cv::Mat& yuvplan)
{
	//BGR.create(yuyv.rows,yuyv.cols,CV_8UC3);
	yuvplan.create(yuyv.rows*3/2,yuyv.cols,CV_8UC1);

	hipMemcpyAsync(tempYUYV, yuyv.data, yuyv.cols*yuyv.rows*2, hipMemcpyHostToDevice,0);
	yuyv2bgr(tempRGB,tempYUYV,yuyv.cols,yuyv.rows,0);
	yuyv2yuvI420_(tempY,tempU,tempV,tempYUYV,yuyv.cols,yuyv.rows,0);
//	hipStreamSynchronize(streamBGR);
//	hipDeviceSynchronize();
	hipMemcpy(BGR.data,tempRGB,yuyv.cols*yuyv.rows*3,hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();
	hipMemcpy(yuvplan.data,tempY,yuyv.cols*yuyv.rows,hipMemcpyDeviceToHost);
	hipMemcpy(yuvplan.data+yuyv.cols*yuyv.rows,tempU,yuyv.cols*yuyv.rows/4,hipMemcpyDeviceToHost);
	hipMemcpy(yuvplan.data+yuyv.cols*yuyv.rows*5/4,tempV,yuyv.cols*yuyv.rows/4,hipMemcpyDeviceToHost);
//	hipStreamSynchronize(NULL);
	//hipStreamDestroy(streamBGR);
	//hipStreamDestroy(streamYUV);
}

void yuyv2BGRI420_MD(cv::Mat& yuyv,cv::Mat& BGR,cv::Mat& yuvplan)
{
	//BGR.create(yuyv.rows,yuyv.cols,CV_8UC3);
	yuvplan.create(yuyv.rows*3/2,yuyv.cols,CV_8UC1);

	int byteCount =  yuyv.cols*yuyv.rows*2;
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*0, yuyv.data + (byteCount>>2)*0, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[0]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*1, yuyv.data + (byteCount>>2)*1, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[1]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*2, yuyv.data + (byteCount>>2)*2, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[2]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*3, yuyv.data + (byteCount>>2)*3, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[3]);

	int byteCount_rgb = yuyv.cols*yuyv.rows*3;
	int byteCount_y = yuyv.cols*yuyv.rows;
	int byteCount_uv = (byteCount_y>>1);
	yuyv2yuvBGRI420_(tempRGB + (byteCount_rgb>>2)*0,
			tempY+(byteCount_y>>2)*0,
			tempU+(byteCount_uv>>2)*0,
			tempV+(byteCount_uv>>2)*0,
			tempYUYV + (byteCount>>2)*0,
			yuyv.cols, (yuyv.rows>>2), streamBGR[0]);
	yuyv2yuvBGRI420_(tempRGB + (byteCount_rgb>>2)*1,
				tempY+(byteCount_y>>2)*1,
				tempU+(byteCount_uv>>2)*1,
				tempV+(byteCount_uv>>2)*1,
				tempYUYV + (byteCount>>2)*1,
				yuyv.cols, (yuyv.rows>>2), streamBGR[1]);
	yuyv2yuvBGRI420_(tempRGB + (byteCount_rgb>>2)*2,
				tempY+(byteCount_y>>2)*2,
				tempU+(byteCount_uv>>2)*2,
				tempV+(byteCount_uv>>2)*2,
				tempYUYV + (byteCount>>2)*2,
				yuyv.cols, (yuyv.rows>>2), streamBGR[2]);
	yuyv2yuvBGRI420_(tempRGB + (byteCount_rgb>>2)*3,
				tempY+(byteCount_y>>2)*3,
				tempU+(byteCount_uv>>2)*3,
				tempV+(byteCount_uv>>2)*3,
				tempYUYV + (byteCount>>2)*3,
				yuyv.cols, (yuyv.rows>>2), streamBGR[3]);

	hipMemcpyAsync(BGR.data + (byteCount_rgb>>2)*0, tempRGB + (byteCount_rgb>>2)*0, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[0]);
	hipMemcpyAsync(BGR.data + (byteCount_rgb>>2)*1, tempRGB + (byteCount_rgb>>2)*1, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[1]);
	hipMemcpyAsync(BGR.data + (byteCount_rgb>>2)*2, tempRGB + (byteCount_rgb>>2)*2, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[2]);
	hipMemcpyAsync(BGR.data + (byteCount_rgb>>2)*3, tempRGB + (byteCount_rgb>>2)*3, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[3]);

	hipMemcpyAsync(yuvplan.data + (byteCount_y>>2)*0, tempY + (byteCount_y>>2)*0, (byteCount_y>>2), hipMemcpyDeviceToHost, streamBGR[0]);
	hipMemcpyAsync(yuvplan.data + (byteCount_y>>2)*1, tempY + (byteCount_y>>2)*1, (byteCount_y>>2), hipMemcpyDeviceToHost, streamBGR[1]);
	hipMemcpyAsync(yuvplan.data + (byteCount_y>>2)*2, tempY + (byteCount_y>>2)*2, (byteCount_y>>2), hipMemcpyDeviceToHost, streamBGR[2]);
	hipMemcpyAsync(yuvplan.data + (byteCount_y>>2)*3, tempY + (byteCount_y>>2)*3, (byteCount_y>>2), hipMemcpyDeviceToHost, streamBGR[3]);

	unsigned char *pu = yuvplan.data+yuyv.cols*yuyv.rows;
	hipMemcpyAsync(pu + (byteCount_uv>>2)*0, tempU + (byteCount_uv>>2)*0, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[0]);
	hipMemcpyAsync(pu + (byteCount_uv>>2)*1, tempU + (byteCount_uv>>2)*1, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[1]);
	hipMemcpyAsync(pu + (byteCount_uv>>2)*2, tempU + (byteCount_uv>>2)*2, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[2]);
	hipMemcpyAsync(pu + (byteCount_uv>>2)*3, tempU + (byteCount_uv>>2)*3, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[3]);

	unsigned char *pv = yuvplan.data+yuyv.cols*yuyv.rows*5/4;
	hipMemcpyAsync(pv + (byteCount_uv>>2)*0, tempV + (byteCount_uv>>2)*0, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[0]);
	hipMemcpyAsync(pv + (byteCount_uv>>2)*1, tempV + (byteCount_uv>>2)*1, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[1]);
	hipMemcpyAsync(pv + (byteCount_uv>>2)*2, tempV + (byteCount_uv>>2)*2, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[2]);
	hipMemcpyAsync(pv + (byteCount_uv>>2)*3, tempV + (byteCount_uv>>2)*3, (byteCount_uv>>2), hipMemcpyDeviceToHost, streamBGR[3]);
}

void yuyv2BGR(cv::Mat& yuyv,cv::Mat& BGR)
{
	int byteCount =  yuyv.cols*yuyv.rows*2;
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*0, yuyv.data + (byteCount>>2)*0, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[0]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*1, yuyv.data + (byteCount>>2)*1, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[1]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*2, yuyv.data + (byteCount>>2)*2, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[2]);
	hipMemcpyAsync(tempYUYV + (byteCount>>2)*3, yuyv.data + (byteCount>>2)*3, (byteCount>>2), hipMemcpyHostToDevice, streamBGR[3]);

	int byteCount_rgb = yuyv.cols*yuyv.rows*3;

	yuyv2bgr(tempRGB + (byteCount_rgb>>2)*0,
			tempYUYV + (byteCount>>2)*0,
			yuyv.cols, (yuyv.rows>>2), streamBGR[0]);
	yuyv2bgr(tempRGB + (byteCount_rgb>>2)*1,
				tempYUYV + (byteCount>>2)*1,
				yuyv.cols, (yuyv.rows>>2), streamBGR[1]);
	yuyv2bgr(tempRGB + (byteCount_rgb>>2)*2,
				tempYUYV + (byteCount>>2)*2,
				yuyv.cols, (yuyv.rows>>2), streamBGR[2]);
	yuyv2bgr(tempRGB + (byteCount_rgb>>2)*3,
				tempYUYV + (byteCount>>2)*3,
				yuyv.cols, (yuyv.rows>>2), streamBGR[3]);

	hipMemcpyAsync(pRGALockMem + (byteCount_rgb>>2)*0, tempRGB + (byteCount_rgb>>2)*0, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[0]);
	hipMemcpyAsync(pRGALockMem + (byteCount_rgb>>2)*1, tempRGB + (byteCount_rgb>>2)*1, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[1]);
	hipMemcpyAsync(pRGALockMem + (byteCount_rgb>>2)*2, tempRGB + (byteCount_rgb>>2)*2, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[2]);
	hipMemcpyAsync(pRGALockMem + (byteCount_rgb>>2)*3, tempRGB + (byteCount_rgb>>2)*3, (byteCount_rgb>>2), hipMemcpyDeviceToHost, streamBGR[3]);

	memcpy(BGR.data, pRGALockMem, BGR.rows*BGR.cols*3);
}

