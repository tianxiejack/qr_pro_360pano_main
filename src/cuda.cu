//#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
//#include "device_launch_parameters.h"

#define clip(minv, maxv, value)  ((value)<minv) ? minv : (((value)>maxv) ? maxv : (value))


__global__ void kernel_scale(
		float *scale, const unsigned char *src, int src_Width, int src_Height, int scale_step)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	int c_step=((src_Width)>>scale_step);
	int sx = (x<<scale_step);
	int sy = (y<<scale_step);

	int src_step = src_Width*4;

	const unsigned char *pS;

	if(x >= (src_Width>>scale_step) || y >= (src_Height>>scale_step))
		return;

	{
		unsigned char R, G, B;
		pS = src + (sy*src_step + sx*4);
		B = pS[0] ;//=  128;//clip(0, 255, (src[src_y*src_step + src_x*3] * exposure[y*dst_Width + x]));
		G = pS[1];// = 128;//clip(0, 255, (src[src_y*src_step + src_x*3 +1] * exposure[y*dst_Width + x]));
		R = pS[2];// = 128;//clip(0, 255, (src[src_y*src_step + src_x*3 +2] * exposure[y*dst_Width + x]));
		//if((y&c_mask)==0 && (x&c_mask)==0)
		{
			scale[y*c_step+x] = 0.299*R + 0.587*G + 0.114*B;
		}
	}
}

#define DESCALE(x, n)    (((x) + (1 << ((n)-1)))>>(n))
#define COEFFS_0 		(22987)
#define COEFFS_1 		(-11698)
#define COEFFS_2 		(-5636)
#define COEFFS_3 		(29049)
__global__ void kernel_uyvy2bgr(
	unsigned char *dst, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= (width>>1) || y >= height)
		return;

	{
	int Y1, Y2, U, V;
	int r, g, b;

	int si = y*width*2 + x*4;
	int di = y*width*3 + x*6;

	U =  src[si+0];
	Y1 = src[si+1];
	V =  src[si+2];
	Y2 = src[si+3];

	b = DESCALE((U - 128)*COEFFS_3, 14);
	g = DESCALE((U - 128)*COEFFS_2 + (V - 128)*COEFFS_1, 14);
	r = DESCALE((V - 128)*COEFFS_0, 14);

	dst[di+0] = clip(0, 255, Y1 + b);//B
	dst[di+1] = clip(0, 255, Y1 + g);//G
	dst[di+2] = clip(0, 255, Y1 + r);//R
	dst[di+3] = clip(0, 255, Y2 + b);//B
	dst[di+4] = clip(0, 255, Y2 + g);//G
	dst[di+5] = clip(0, 255, Y2 + r);//R
	}
}

__global__ void kernel_yuyv2bgr(
	unsigned char *dst, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= (width>>1) || y >= height)
		return;

	{
		int Y1, Y2, U, V;
		int r, g, b;

		int si = y*width*2 + x*4;
		int di = y*width*3 + x*6;

		Y1 =  src[si+0];
		U = src[si+1];
		Y2 =  src[si+2];
		V = src[si+3];

		b = DESCALE((U - 128)*COEFFS_3, 14);
		g = DESCALE((U - 128)*COEFFS_2 + (V - 128)*COEFFS_1, 14);
		r = DESCALE((V - 128)*COEFFS_0, 14);

		dst[di+0] = clip(0, 255, Y1 + b);//B
		dst[di+1] = clip(0, 255, Y1 + g);//G
		dst[di+2] = clip(0, 255, Y1 + r);//R
		dst[di+3] = clip(0, 255, Y2 + b);//B
		dst[di+4] = clip(0, 255, Y2 + g);//G
		dst[di+5] = clip(0, 255, Y2 + r);//R
	}
}

__device__ void deInterlace(unsigned char top, unsigned char bot, unsigned char mid, unsigned char *dst)
{
    *dst = clip(0, 255, (int)top + bot + mid- min(min(top, bot), mid) - max(max(top, bot), mid));
}

__device__ void deInterlaceUV(int top, int bot, int mid, unsigned char *dst)
{
    *dst = clip(0, 255, top + bot + mid- min(min(top, bot), mid) - max(max(top, bot), mid));
}

__device__ void deInterlaceY(unsigned char top[], unsigned char bot[], unsigned char mid, unsigned char *dst)
{
	const int thred = 50;
	int grd = abs(top[0] - mid) + abs(top[1] - mid) + abs(top[2] - mid) +
					   abs(bot[0] - mid) + abs(bot[1] - mid) + abs(bot[2] - mid);

	if(grd > thred){
		int grda = abs(top[0] - bot[2]);
		int grdb = abs(top[1] - bot[1]);
		int grdc = abs(top[2] - bot[0]);

		if( (grda < grdb) && (grda < grdc) )
		{
			*dst = top[0] + bot[2] + mid -min(min(top[0], bot[2]), mid) - max(max(top[0], bot[2]), mid);//medthr(a_1, b1, d);
		}
		else if( (grdc < grda) && (grdc < grdb) )
		{
			*dst = top[2] + bot[0] + mid - min(min(top[2], bot[0]), mid) - max(max(top[2], bot[0]), mid);//medthr(a1, b_1, d);
		}
		else
		{
			*dst = top[1] + bot[1] + mid- min(min(top[1], bot[1]), mid) - max(max(top[1], bot[1]), mid);//medthr(a, b, d);
		}
	}
	else
	{
		*dst = mid;
	}
}

#if 0
__global__ void kernel_dei( unsigned char *src, int width, int height)
{
	//dei
	int x = blockIdx.x * (blockDim.x ) + (threadIdx.x );
	int y = blockIdx.y * (blockDim.y )  + (threadIdx.y);

	if((x+1) >= width || (y+1) >= height)
		return;

	//return;
	width = width * 2;
	x = x*2;//x*4;
	y = y*2;

	int offset_top = (y + 0) * width + x;     //top point
    int offset_mid = (y + 1) * width + x;     //mid point
    int offset_bot = (y + 2) * width + x;     //bot point

    unsigned char topY[3], botY[3], midY;
    unsigned char topUV, botUV, midUV;

    topUV = src[offset_top + 0];       //U or V
    botUV = src[offset_bot + 0];
    midUV = src[offset_mid + 0];

    topY[0] = src[offset_top - 1];       //Y
    topY[1] = src[offset_top + 1];
    topY[2] = src[offset_top + 3];
    botY[0] = src[offset_bot - 1];
    botY[1] = src[offset_bot + 1];
    botY[2] = src[offset_bot + 3];
    midY = src[offset_mid + 1];

	//select the middle pixel to replace the mid one
    //deInterlaceUV(topUV, botUV , midUV, &src[offset_mid + 0]);
    deInterlaceY(topY, botY , midY, &src[offset_mid + 1]);

}
#else
__global__ void kernel_dei(unsigned char *src, int width, int height)
{
	//dei
	int x = blockIdx.x * (blockDim.x ) + (threadIdx.x );
	int y = blockIdx.y * (blockDim.y )  + (threadIdx.y);

	if((x+1) >= width || (y+1) >= height)
		return;

	//return;
	width = width * 2;
	x = x * 4;
	y = y*2;

	int offset_top = (y + 0) * width + x;     //top point
    int offset_mid = (y + 1) * width + x;     //mid point
    int offset_bot = (y + 2) * width + x;     //bot point

    unsigned char top[4], bot[4], mid[4];

    top[0] = src[offset_top + 0];       //U
    bot[0] = src[offset_bot + 0];
    mid[0] = src[offset_mid + 0];

    top[1] = src[offset_top + 1];       //Y
    bot[1] = src[offset_bot + 1];
    mid[1] = src[offset_mid + 1];

    top[2] = src[offset_top + 2];       //V
    bot[2] = src[offset_bot + 2];
    mid[2] = src[offset_mid + 2];

    top[3] = src[offset_top + 3];       //Y
    bot[3] = src[offset_bot + 3];
    mid[3] = src[offset_mid + 3];

	//select the middle pixel to replace the mid one
//    deInterlace(top[0], bot[0] , mid[0], &src[offset_mid + 0]);
    deInterlace(top[1], bot[1] , mid[1], &src[offset_mid + 1]);
//    deInterlace(top[2], bot[2] , mid[2], &src[offset_mid + 2]);
    deInterlace(top[3], bot[3] , mid[3], &src[offset_mid + 3]);

}
#endif

extern "C" int uyvydei_(
	unsigned char *dst,
	int width, int height)
{
	dim3 block((width/2+31)/32,(height/2+31)/32);
	dim3 thread(32, 32);
	//dim3 block((dst_Width+127)/128,(dst_Height+127)/128);
	//dim3 thread(128, 128);
	kernel_dei<<<block, thread>>>(dst, width, height);

	return 0;
}

#if 1
extern "C" int uyvy2bgr_(
	unsigned char *dst, const unsigned char *src,
	int width, int height, hipStream_t stream)
{
	//dim3 block((width/2+15)/16, (height+63)/64);
	//dim3 thread(16, 64);
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);
	//dim3 block((width/2+127)/128,(height+127)/128);
	//dim3 thread(128, 128);

	//kernel_uyvy2bgr_and_sphere_tp_erect<<<block, thread>>>(dst, src, width, height);
	kernel_uyvy2bgr<<<block, thread, 0, stream>>>(dst, src, width, height);

	return 0;
}
extern "C" int yuyv2bgr_(
	unsigned char *dst, const unsigned char *src,
	int width, int height, hipStream_t stream)
{
	//dim3 block((width/2+15)/16, (height+63)/64);
	//dim3 thread(16, 64);
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);
	//dim3 block((width/2+127)/128,(height+127)/128);
	//dim3 thread(128, 128);

	//kernel_uyvy2bgr_and_sphere_tp_erect<<<block, thread>>>(dst, src, width, height);
	kernel_yuyv2bgr<<<block, thread, 0, stream>>>(dst, src, width, height);

	return 0;
}
#else
extern "C" int uyvy2bgr_(
	unsigned char *dst, const unsigned char *src,
	int width, int height)
{
	dim3 block((width/2+15)/16, (height+63)/64);
	dim3 thread(16, 64);

	cudaChannelFormatDesc desc = cudaCreateChannelDesc<unsigned char>();
	cudaBindTexture2D(NULL, texIn, src, desc, width*2, height, width*2);

	kernel_uyvy2bgr<<<block, thread>>>(dst, src, width, height);

	cudaUnbindTexture(texIn);

	return 0;
}
#endif

extern "C" int kernel_scale_(
		float *scale, const unsigned char *src,
	int width, int height, int scale_step)
{
	dim3 block(((width>>scale_step)+31)/32,((height>>scale_step)+31)/32);
	dim3 thread(32, 32);
	//dim3 block((dst_Width+127)/128,(dst_Height+127)/128);
	//dim3 thread(128, 128);

	kernel_scale<<<block, thread>>>(scale, src, width, height, scale_step);

	return 0;
}

__global__ void kernel_yuyv2bgr_ext(
	unsigned char *dst, const unsigned char *src, unsigned char *gray,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= (width>>1) || y >= height)
		return;

	{
		int Y1, Y2, U, V;
		int r, g, b;

		int si = y*width*2 + x*4;
		int di = y*width*3 + x*6;

		Y1 =  src[si+0];
		U = src[si+1];
		Y2 =  src[si+2];
		V = src[si+3];

		gray[y*width + x*2 + 0] = Y1;
		gray[y*width + x*2 + 1] = Y2;

		b = DESCALE((U - 128)*COEFFS_3, 14);
		g = DESCALE((U - 128)*COEFFS_2 + (V - 128)*COEFFS_1, 14);
		r = DESCALE((V - 128)*COEFFS_0, 14);

		dst[di+0] = clip(0, 255, Y1 + b);//B
		dst[di+1] = clip(0, 255, Y1 + g);//G
		dst[di+2] = clip(0, 255, Y1 + r);//R
		dst[di+3] = clip(0, 255, Y2 + b);//B
		dst[di+4] = clip(0, 255, Y2 + g);//G
		dst[di+5] = clip(0, 255, Y2 + r);//R
	}
}

extern "C" int yuyv2bgr_ext_(
	unsigned char *dst, const unsigned char *src, unsigned char *gray,
	int width, int height, hipStream_t stream)
{
	//dim3 block((width/2+15)/16, (height+63)/64);
	//dim3 thread(16, 64);
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);
	//dim3 block((width/2+127)/128,(height+127)/128);
	//dim3 thread(128, 128);

	//kernel_uyvy2bgr_and_sphere_tp_erect<<<block, thread>>>(dst, src, width, height);
	kernel_yuyv2bgr_ext<<<block, thread, 0, stream>>>(dst, src, gray, width, height);

	return 0;
}


__global__ void kernel_yuyv2gray(
	unsigned char *dst, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if(x >= width || y >= height)
		return;

	dst[y*width + x] = src[y*width*2 + x*2];
}

extern "C" int yuyv2gray_(
	unsigned char *dst, const unsigned char *src,
	int width, int height, hipStream_t stream)
{
	dim3 block((width+31)/32,(height+31)/32);
	dim3 thread(32, 32);

	kernel_yuyv2gray<<<block, thread, 0, stream>>>(dst, src, width, height);

	return 0;
}

__global__ void kernel_yuyv2yuvplan(
	unsigned char *dsty, unsigned char *dstu, unsigned char *dstv, const unsigned char *src,
	int width, int height)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int uvwidth = (width>>1);

	if(x >= (width>>1) || y >= height)
		return;

	dsty[y*width + 2*x] 		= 		src[y*width*2 + x*4];
	dstu[y*uvwidth + x] 	= 		src[y*width*2 + x*4+1];
	dsty[y*width + 2*x+1] = 		src[y*width*2 + x*4+2];
	dstv[y*uvwidth + x] 		= 		src[y*width*2 + x*4+3];
}

extern "C" int yuyv2yuvplan_(
	unsigned char *dsty, unsigned char *dstu,unsigned char *dstv,const unsigned char *src,
	int width, int height, hipStream_t stream)
{
	dim3 block((width/2+31)/32,(height+31)/32);
	dim3 thread(32, 32);

	kernel_yuyv2yuvplan<<<block, thread, 0, stream>>>(dsty,dstu,dstv, src, width, height);

	return 0;
}



